#include "PinHoleGPU.cuh"

namespace camera
{
	__device__ Ray PinholeGPU::getRayTo(Vector2 pictureLocation) const
	{
		return Ray(origin, rayDirection(pictureLocation));
	}

	__device__ Vector3 PinholeGPU::rayDirection(Vector2 v) const
	{
		return onb * Vector3(v.x, v.y, -distance);
	}

	__device__ PinholeGPU::PinholeGPU(Vector3 origin, Vector3 lookAt, Vector3 up, float distance)
		: onb(origin, lookAt, up), origin(origin), distance(distance) {}

	__device__ PinholeGPU::PinholeGPU() {}

	PinholeGPU* PinholeGPU::GpuAlloc(Vector3 origin, Vector3 lookAt, Vector3 up, float distance) {
		PinholeGPU obj = PinholeGPU(origin, lookAt, up, distance);
		PinholeGPU* d_ptr;

		hipMalloc((void**)&d_ptr, sizeof(PinholeGPU));
		hipMemcpy(d_ptr, &obj, sizeof(PinholeGPU), hipMemcpyHostToDevice);

		return d_ptr;
	}
}
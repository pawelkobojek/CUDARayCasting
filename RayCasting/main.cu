#include "hip/hip_runtime.h"
#include <SDL.h>
#include <cstdlib>
#include <iostream>

#include "MainWindow.cuh"
//#include "RayCastingGPU.h"
//#include "RayCastingCPU.h"
#include "Engine/RayTracerCPU.cuh"
#include "Engine/RayTracerGPU.cuh"
#include "Geometries/Sphere.cuh"
#include "Geometries/SphereGPU.cuh"
#include "Geometries/Plane.cuh"
#include "Camera/PinHole.cuh"
#include "Camera/PinholeGPU.cuh"
#include "Common/ColorRGB.cuh"
#include "Materials/IMaterial.cuh"
#include "Materials/Phong.cuh"
#include "Materials/PerfectDiffuse.cuh"

#define BACKGROUND_COLOR COLOR(200,200,255,255)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;
using namespace engine;
using namespace geometries;
using namespace camera;
using namespace materials;

__global__ void rayTraceKernelTest(uint32_t* image, int width, int height) {
	//int X = threadIdx.x;// / WIDTH;
	//int Y = threadIdx.y;// %  HEIGHT;

	//for(int X = 0; X < width; X++) {
	//for (int Y = 0; Y < height; Y++)
	//{
	//image[PIXEL(X, Y)] = 0xFF0000;//ColorRGB(1.0f, 0.0f, 0.0f);
	//}
	//}
}

__global__ void rayTraceKernel(RayTracerGPU rayTracerGPU, uint32_t* image, World* world, PinholeGPU* camera, int width, int height) {
	rayTracerGPU.rayTrace(image, world, camera, width, height, blockIdx.x, threadIdx.x);
	//image[blockIdx.x, threadIdx.x]=0xff000000;
	//int X = blockIdx.x;
	//int Y = threadIdx.x;

	//for(int X = 0; X < width; X++) {
	//for (int Y = 0; Y < height; Y++)
	//{
	//image[PIXEL(X, Y)] = 0xFF0000;//ColorRGB(1.0f, 0.0f, 0.0f);
	//}
	//}

	//int x = threadIdx.x, y = threadIdx.y;
	//Vector2 pcoord = Vector2(
	//((x + 0.5f) / (float)width) * 2 - 1,
	//((y + 0.5f) / (float)height) * 2 - 1);

	//Ray ray = camera->getRayTo(pcoord);

	//shadeRayGPU(world, ray, &image[PIXEL(x, y)]);
}

int main(int argc, char** argv)
{
	MainWindow window = MainWindow();
	uint32_t* data = new uint32_t[TEXTURE_SIZE];
	uint32_t* d_data;
	RayTracerCPU rayTracerCPU;
	RayTracerGPU rayTracerGPU;

	SphereGPU* spheres = new SphereGPU[3];
	spheres[0] = SphereGPU(Vector3(-4.0f, 0, 0), 2, Phong(ColorRGB::red, 0.5f, 1.f, 30.f));
	spheres[1] = SphereGPU(Vector3(4.0f, 0, 0), 2, Phong(ColorRGB::red, 0.5f, 1.f, 30.f));
	spheres[2] = SphereGPU(Vector3(0, 0, 3.0f), 2, Phong(ColorRGB::red, 0.5f, 1.f, 30.f));


	IMaterial* redMat = new Phong(ColorRGB::red, 0.5f, 1.f, 30.f);
	IMaterial* greenMat = new Phong(ColorRGB::green, 0.5f, 1.f, 30.f);
	IMaterial* blueMat = new Phong(ColorRGB::blue, 0.5f, 1.f, 30.f);
	IMaterial* grayMat = new Phong(ColorRGB::gray, 0.5f, 1.f, 30.f);

	World world = World(BACKGROUND_COLOR, 5, 5);
	world.add(new Sphere(Vector3(-4.0f, 0, 0), 2, redMat));
	world.add(new Sphere(Vector3(4.0f, 0, 0), 2, greenMat));
	world.add(new Sphere(Vector3(0, 0, 3.0f), 2, blueMat));
	world.add(new Plane(Vector3(0, -2, 0), Vector3(0, 1, 0), grayMat));

	world.addLight(new PointLight(Vector3(0, 5, -5), ColorRGB::white));

	ICamera* cam = new Pinhole(Vector3(0, 1, -8), Vector3(0, 0, 0), Vector3(0, -1, 0), 1);
	PinholeGPU pincam = PinholeGPU(Vector3(0, 1, -8), Vector3(0, 0, 0), Vector3(0, -1, 0), 1);
	RayTracerGPU rtGpu = RayTracerGPU();
	RayTracerGPU* d_rtGpu;
	PinholeGPU* d_cam = PinholeGPU::GpuAlloc(Vector3(0, 1, -8), Vector3(0, 0, 0), Vector3(0, -1, 0), 1);
	hipMalloc((void**)&d_data, TEXTURE_SIZE * sizeof(uint32_t));
	hipMemcpy(d_data, data, TEXTURE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_rtGpu, sizeof(RayTracerGPU));
	hipMemcpy(d_rtGpu, &rtGpu, sizeof(RayTracerGPU), hipMemcpyHostToDevice);

	cout << "RayTracerGPU: " << sizeof(RayTracerGPU) << "\nuint32_t: " << sizeof(uint32_t*) 
		<< "\nWorld: " << sizeof(World) << "\nPinholeGPU: " << sizeof(PinholeGPU) << "\nTotal size of parameteres: "
		<< sizeof(RayTracerGPU) + sizeof(uint32_t*) + sizeof(World) + sizeof(PinholeGPU) << endl;

	World* d_world_ptr = World::GpuAllocFrom(&world);

	bool quit = false;
	while (!quit)
	{
		window.fpsLimiter.startFrame();

		while (SDL_PollEvent(&window.event)) quit = window.handleEvents();

		if (window.usesGPU()) {
			rayTraceKernel<<<WIDTH,HEIGHT>>>(rayTracerGPU, d_data, d_world_ptr, d_cam, WIDTH, HEIGHT);
			gpuErrchk(hipPeekAtLastError());
			hipDeviceSynchronize();
			hipMemcpy(data, d_data, TEXTURE_SIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);
		}
		else{ 
			rayTracerCPU.rayTrace(data, &world, cam, WIDTH, HEIGHT); //rayCastingCPU(data, WIDTH, HEIGHT);
		}

		window.update(data);
		window.fpsLimiter.endFrame();
		window.fpsLimiter.delay();
		window.updateWindowTitle();
	}
	delete redMat;
	delete greenMat;
	delete blueMat;
	delete grayMat;
	delete cam;
	delete[] data;

	hipFree(d_rtGpu);
	hipFree(d_cam);
	hipFree(d_data);
	World::GpuFree(d_world_ptr);

	return 0;
}
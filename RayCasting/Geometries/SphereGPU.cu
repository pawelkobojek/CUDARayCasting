#include "hip/hip_runtime.h"
#include "SphereGPU.cuh"
#include <cmath>

namespace geometries
{
	__host__ __device__ SphereGPU::SphereGPU(const Vector3 center, const float radius, Phong material)
		: material(material), center(center), radius(radius) {}

	__host__ __device__ SphereGPU::SphereGPU() {}


	__host__ __device__ bool SphereGPU::hitTest(const Ray ray, float* minDistance, Vector3* outNormal) const
	{
		float t;
		Vector3 distance = ray.origin - center;
		float a = ray.direction.lengthSq();
		float b = (distance * 2).dot(ray.direction);
		float c = distance.lengthSq() - radius * radius;
		float disc = b*b - 4 * a*c;

		if (disc < 0) return false;
		float discSq = sqrtf(disc);
		float denom = 2 * a;

		t = (-b - discSq) / denom;
		if (t < 0.0001)
		{
			t = (-b + discSq) / denom;
		}
		if (t < 0.0001)
		{
			return false;
		}

		Vector3 hitPoint = (ray.origin + ray.direction * t);
		*outNormal = (hitPoint - center).normalized();
		*minDistance = t;
		return true;
	}
}